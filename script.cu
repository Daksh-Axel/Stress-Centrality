#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>
#define inf 1000000000
using namespace std;



__device__ void recover_path(int* par,int s,int e, int *path, int *bc,int n){
    if(e==s){
        for(int i=1;i<path[0];i++){
            if(path[i]==e) break;
            bc[path[i]]++;
        }
        return;
    }

    for(int v=1;v<=par[e*n];v++){
        int p=par[e*n+v];
        path[++path[0]]=p;
        recover_path(par,s,p,path,bc,n);
        path[0]--;
    }
}



__device__ void STRESS_CENTRALITY(int src,int n, int *adj,int *stress_cen,int uver,int vver){

    int *par = new int[n*(n+1)];
    int *dist= new int[n];
    for(int i=0;i<n;i++){
        dist[i]=inf;
        par[i*n]=0;
    }
    int *q=new int[n];
    int wcnt=0,rcnt=0;
    
    q[wcnt++]=src;
    dist[src]=0;

    while(rcnt<wcnt){
        int u=q[rcnt++];
        for(int ver=1;ver<=adj[u*n]+1;ver++){
            int v;
            if(ver==adj[u*n]+1){
                if(u==uver) v=vver;
                else if(u==vver) v=uver;
                else break;
            }
            else v=adj[u*n+ver];
            if(dist[v]>dist[u]+1){
                dist[v] = dist[u]+1;
                par[v*n] = 0;
                par[v*n + par[v*n]+1] = u; par[v*n]++;
                q[wcnt++] = v;
            }
            else if(dist[v]==dist[u]+1){
                par[v*n + par[v*n]+1] = u; par[v*n]++;
                
            }
        }
    }
    for(int dst=0;dst<n;dst++){
        if(src!=dst && dist[dst]!=inf){
            int *SC = new int[n]; 
            for(int i=0;i<n;i++) SC[i]=0;
            int *path = new int[n];
            path[0]=0;
            recover_path(par,src,dst,path,SC,n);
            for(int v=0;v<n;v++){
                stress_cen[v]+=SC[v];
            }
            delete[] SC;
            delete[] path;
        }
    }
    delete[] par;
    delete[] dist;
    delete[] q;
    
}
__global__ void SC_kernel(int n,int *adj,int *SC,int u,int v,int *min_SC,int *opt_edge){
    int src = blockIdx.x*blockDim.x + threadIdx.x;
    int *temp = new int[n];
    
    for(int node=0;node<n;node++){
        temp[node]=0;
        SC[node]=0;
    }
    STRESS_CENTRALITY(src,n,adj,temp,u,v);
     __syncthreads();
   
    for(int node = 0;node<n;node++){
        atomicAdd(&SC[node], temp[node]);
    }
     __syncthreads();
    if(src==0){
        for(int node=0;node<n;node++){
          SC[node]/=2;
          if(min_SC[node]>SC[node]){
              min_SC[node]=SC[node];
              opt_edge[node]=u*n+v;
          }
        }
    }
    delete[] temp;
}

int main()
{
    freopen("test.edgelist","r",stdin);
    freopen("op.txt","w+",stdout);
    
    int n,m;
    cin>>n>>m;
    int adj[n*(n+1)];
    int SC[n];
    set<pair<int,int>> edges;
    
    for(int i=0;i<n;i++){
        adj[i*n]=0;
        SC[i]=0;
    }
    for(int i=0;i<m;i++){
        int u,v;
        cin>>u>>v;
        adj[u*n+adj[u*n]+1]=v; adj[u*n]++;
        adj[v*n+adj[v*n]+1]=u; adj[v*n]++;
        edges.insert({u,v});
    }
 
    int *d_adj;
    hipMalloc(&d_adj,n*(n+1)*sizeof(int));
    hipMemcpy(d_adj,adj,n*(n+1)*sizeof(int),hipMemcpyHostToDevice);
 
    int *d_SC;
    hipMalloc(&d_SC,n*sizeof(int));
    hipMemcpy(d_SC,SC,n*sizeof(int),hipMemcpyHostToDevice);
 
    int max_SC[n];for(int i=0;i<n;i++) max_SC[i] = inf;
 
    int *d_min_SC;
    hipMalloc(&d_min_SC,n*sizeof(int));
    hipMemcpy(d_min_SC,max_SC,n*sizeof(int),hipMemcpyHostToDevice);
    
    int *opt_edge;
    hipMalloc(&opt_edge,n*sizeof(int));
    hipMemcpy(opt_edge,SC,n*sizeof(int),hipMemcpyHostToDevice);
    
    SC_kernel<<<1, n>>>(n,d_adj,d_SC,-1,-1,d_min_SC,opt_edge);
    hipDeviceSynchronize();
    hipMemcpy(SC,d_min_SC,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Initial Stress Centrality:\n");
    for(int i=0;i<n;i++) printf("%d ",SC[i]);
    printf("\n");

 
    for(int u=0;u<n;u++){
        for(int v=u+1;v<n;v++){
            if(edges.find({u,v})!=edges.end()) continue;
            SC_kernel<<<1, n>>>(n,d_adj,d_SC,u,v,d_min_SC,opt_edge);
            hipError_t hipError_t = hipGetLastError();
            if (hipError_t != hipSuccess) {
                const char* errorMessage = hipGetErrorString(hipError_t);
                printf("CUDA error: %s\n", errorMessage);
                break;
            }
            hipDeviceSynchronize();
        }
     }
    
    hipMemcpy(SC,d_min_SC,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Minimum Stress Centrality:\n");
    for(int i=0;i<n;i++) printf("%d ",SC[i]);
 
    int oe[n];
    hipMemcpy(oe,opt_edge,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Optimal Edges for respective nodes:\n");
    for(int node=0;node<n;node++){
        int a=oe[node]%n,b=oe[node]/n;
        printf("Node: %d ==> (%d,%d)\n",node,a,b);
    }

    return 0;
}
